//
// Created by progamers on 9/25/25.
//
#include "gpu/algorithm.h"
#include "gpu/kernel.h"
#include <hip/hip_runtime.h>

namespace gpu {
void find_path(hipSurfaceObject_t array, position* path, std::vector<position> path_cpu,
			   type width, type height, bool* flag, hipStream_t stream) {
	void* kernel_args[] = {&array, &width, &height, &flag};
	hipLaunchCooperativeKernel(reinterpret_cast<const void*>(simple_path_finding), gridDim, blockDim, kernel_args, 0, stream);
}

} // namespace gpu